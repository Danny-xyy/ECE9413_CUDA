#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudastart.h"

//CPU对照组，用于对比加速比
void sumMatrix2DonCPU(float * MatA, float * MatB, float * MatC, int nx, int ny)
{
    float* a = MatA;
    float* b = MatB;
    float* c = MatC;
    for (int j = 0; j < ny; j++)
    {
        for(int i = 0; i < nx; i++)
        {
            c[i] = a[i] + b[i];
        }
        c += nx;
        b += nx;
        a += nx;
    }
}

__global__ void sumMatrix(float * MatA, float * MatB, float * MatC, int nx, int ny){
    int ix = threadIdx.x + blockDim.x*blockIdx.x;
    int iy = threadIdx.y + blockDim.y*blockIdx.y;
    int idx = ix + iy * ny;
    if (ix < nx && iy < ny){
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

